#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>

#define BLOCK_SIZE 256

#define LOW (-1.0)
#define HIGH 1.0

# define CUDA_SAFE_CALL(call) {                                                 \
    hipError_t err = call;                                                       \
    if (hipSuccess != err) {                                                   \
        fprintf(stderr, "error %d: Cuda error in file '%s' in line %i : %s.\n", \
                err, __FILE__, __LINE__, hipGetErrorString(err));              \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}

float float_rand(float high, float low);
void cpu_scan(float *output, float *input, int vector_size);
void gpu_scan(float *output, float *input, int vector_size);
__global__ void gpu_scan_kernel(float *d_input, int vector_size, float *d_block_tail);
__global__ void gpu_broadcast_kernel(float *d_input, int vector_size, float *d_block_tail);


int main()
{
    // initialize the vector to be operated on
    int vector_size = 1073741824;
    float *input = (float *)malloc(vector_size * sizeof(float));
    float *output_cpu = (float *)malloc(vector_size * sizeof(float));
    float *output_gpu = (float *)malloc(vector_size * sizeof(float));
    if (input == NULL || output_cpu == NULL || output_gpu == NULL)
    {
        fprintf(stderr, "error: malloc() failed in file '%s' in line %i.\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    srand(time(0));
    for (int i = 0; i < vector_size; i++)
        input[i] = float_rand(LOW, HIGH);

    // prepare timer
    struct timeval begin, end;
    double elapsed_sec;

    // launch the function using CPU
    printf("CPU start ...\n");
    gettimeofday(&begin, NULL);
    cpu_scan(output_cpu, input, vector_size);
    gettimeofday(&end, 0);
    elapsed_sec = (double)(end.tv_sec - begin.tv_sec) + (double)(end.tv_usec - begin.tv_usec) / 1000000.0;
    printf("cpu_time: %lf sec\n", elapsed_sec);
    
    // launch the function using GPU
    printf("GPU start ...\n");
    gettimeofday(&begin, NULL);
    gpu_scan(output_gpu, input, vector_size);
    gettimeofday(&end, 0);
    elapsed_sec = (double)(end.tv_sec - begin.tv_sec) + (double)(end.tv_usec - begin.tv_usec) / 1000000.0;
    printf("gpu_time: %lf sec\n", elapsed_sec);

    // result checking
    for (int i = 0; i < vector_size; i++)
    {
        if (abs(output_cpu[i] - output_gpu[i]) >= 10.0)
        {
            printf("INCORRECT!\n");
            printf("output_cpu[%d] = %f\n", i, output_cpu[i]);
            printf("output_gpu[%d] = %f\n", i, output_gpu[i]);
            return 0;
        }
    }
    printf("CORRECT!\n");

    free(input);
    free(output_cpu);
    free(output_gpu);
    return 0;
}

float float_rand(float high, float low)
{
    return (float)rand() / (float)RAND_MAX * (high - low) + low;
}

void cpu_scan(float *output, float *input, int vector_size)
{
    output[0] = input[0];
    for (int i = 1; i < vector_size; i++)
        output[i] = output[i - 1] + input[i];
}

void gpu_scan(float *output, float *input, int vector_size)
{
    // allocate input space on the device
    float *d_input;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_input, vector_size * sizeof(float)));
    CUDA_SAFE_CALL(hipMemcpy(d_input, input, vector_size * sizeof(float), hipMemcpyHostToDevice));

    // launch the kernel
    int num_blocks = (vector_size - 1) / BLOCK_SIZE + 1;
    if (num_blocks == 1)
        gpu_scan_kernel<<<num_blocks, BLOCK_SIZE>>>(d_input, vector_size, NULL);
    else
    {
        // allocate space to store the tails of each block
        float *h_block_tail = (float *)malloc((num_blocks - 1) * sizeof(float));
        float *d_block_tail;
        CUDA_SAFE_CALL(hipMalloc((void **)&d_block_tail, (num_blocks - 1) * sizeof(float)));

        // launch the kernel
        gpu_scan_kernel<<<num_blocks, BLOCK_SIZE>>>(d_input, vector_size, d_block_tail);

        // gather the tail result
        CUDA_SAFE_CALL(hipMemcpy(h_block_tail, d_block_tail, (num_blocks - 1) * sizeof(float), hipMemcpyDeviceToHost));

        // recursively call the gpu_scan() on the block_tail array
        gpu_scan(h_block_tail, h_block_tail, num_blocks - 1);

        // broadcast the tail results to all the blocks
        CUDA_SAFE_CALL(hipMemcpy(d_block_tail, h_block_tail, (num_blocks - 1) * sizeof(float), hipMemcpyHostToDevice));
        gpu_broadcast_kernel<<<num_blocks, BLOCK_SIZE>>>(d_input, vector_size, d_block_tail);

        free(h_block_tail);
        CUDA_SAFE_CALL(hipFree(d_block_tail));
    }
    CUDA_SAFE_CALL(hipMemcpy(output, d_input, vector_size * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipFree(d_input));
}

__global__
void gpu_scan_kernel(float *d_input, int vector_size, float *d_block_tail)
{
    // calculate the scan_index for each individual thread
    int scan_index = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float block_input[BLOCK_SIZE];        
    if (scan_index < vector_size)
    {
        // load the input into the shared memory
        block_input[threadIdx.x] = d_input[scan_index];

        // decide the adder according to the growing stride
        for (int stride = 1; stride <= threadIdx.x; stride *= 2)
        {
            __syncthreads();
            float adder = block_input[threadIdx.x - stride];
            __syncthreads();
            block_input[threadIdx.x] += adder;
        }

        // store the result to d_input[]
        d_input[scan_index] = block_input[threadIdx.x];

        // store the tail into d_block_tail[]
        int num_blocks = (vector_size - 1) / BLOCK_SIZE + 1;
        if (blockIdx.x < num_blocks - 1 && threadIdx.x == BLOCK_SIZE - 1)
            d_block_tail[blockIdx.x] = block_input[threadIdx.x];
    }
}

__global__
void gpu_broadcast_kernel(float *d_input, int vector_size, float *d_block_tail)
{
    // calculate the global_index for each thread
    int global_index = blockIdx.x * blockDim.x + threadIdx.x;

    // broadcast
    if (global_index < vector_size && blockIdx.x > 0)
        d_input[global_index] += d_block_tail[blockIdx.x - 1];
}
