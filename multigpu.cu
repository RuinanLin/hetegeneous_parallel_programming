#include "hip/hip_runtime.h"
// Copyright (c) 2020 MIT
// Author: Xuhao Chen
#include <hipcub/hipcub.hpp>
//#include "edgelist.h"
#include "graph_gpu.h"
#include "graph_partition.h"
#include "scheduler.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"

typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;
#include "bs_warp_edge.cuh"
#include "bs_warp_vertex.cuh"
#include <thread>

#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>

/**************************************** Definition of SubGraphs ***************************************/

class Type1SubGraph {
  protected:
    vidType super_num_vertices; // how many vertices in the super graph
    int num_partitions;         // the super graph is partitioned into how many partitions
    int this_partition_idx;     // the index of the partition corresponding to this subgraph
    vidType start_vertex_idx;   // the start index of the vertices in this partition, in the global view
    vidType this_num_vertices;  // how many vertices in this partition
    std::ofstream logFile;      // its own logFile

    std::vector<vidType> edges; // column indices of CSR format
    eidType *row_pointers;      // row pointers of CSR format

  public:
    void init(Graph &g, int num_partitions, int this_partition_idx);
    void destroy();
};

// initialize the subgraph
void Type1SubGraph::init(Graph &g, int num_partitions, int this_partition_idx) :
    super_num_vertices(g.V()), num_partitions(num_partitions), this_partition_idx(this_partition_idx)
{
  // initialize its own logFile
  std::string file_name = "Type1SubGraph_log" + std::to_string(this_partition_idx) + ".txt";
  logFile.open(file_name);
  if (!logFile)
  {
    std::cerr << "Cannot open " << file_name << "\n";
    exit(-1);
  }
  logFile << "logFile created!\nType1SubGraph " << this_partition_idx << " initialization starts ...\n";

  // initialize private variables
  int normal_vertex_number_each_partition = (super_num_vertices - 1) / num_partitions + 1;
  start_vertex_idx = normal_vertex_number_each_partition * this_partition_idx;
  this_num_vertices = (this_partition_idx == num_partitions - 1) ? (super_num_vertices - normal_vertex_number_each_partition * (num_partitions - 1)) : normal_vertex_number_each_partition;
  logFile << "\tPrivate variables initialized!\n";
  logFile << "\t\tsuper_num_vertices = " << super_num_vertices << "\n";
  logFile << "\t\tnum_partitions = " << num_partitions << "\n";
  logFile << "\t\tthis_partition_idx = " << this_partition_idx << "\n";
  logFile << "\t\tstart_vertex_idx = " << start_vertex_idx << "\n";
  logFile << "\t\tthis_num_vertices = " << this_num_vertices << "\n";

  // initialize the "row_pointers"
  logFile << "\t"

  // finish initialization and exit
  logFile << "Initialization succeeded!\n";
}

// destroy the subgraph
void Type1SubGraph::destroy()
{
  logFile << "Start destroying Type1SubGraph " << this_partition_idx << " ...\n";

  // free the "edges_from_outside" and "edges_to_outside"
  logFile << "\tFree the allocated memory ...\n";
  free(edges_from_outside);
  free(row_pointers_for_edges_from_outside);
  free(edges_to_outside);
  free(row_pointers_for_edges_to_outside);
  logFile << "\tAllocated memory freed!\n";

  // close the logFile
  logFile << "Gracefully finishing ...\n";
  logFile.close();
}

/**************************************** Definition of TCSolver ***************************************/

void TCSolver(Graph &g, uint64_t &total, int n_gpus, int chunk_size) {
  // start TCSolver
  std::string file_name = "log.txt";
  std::ofstream logFile(file_name);
  if (!logFile)
  {
    std::cerr << "Cannot open " << file_name << "\n";
    exit(-1);
  }
  logFile << "TCSolver starts ...\n";

  // read important information out from the super graph
  logFile << "Reading input graph ...\n";
  vidType super_graph_vertex_num = g.V();
  logFile << "|V| = " << super_graph_vertex_num << "\n";

  // get the device_count of the system
  logFile << "Looking for devices ...\n";
  int device_count;
  CUDA_SAFE_CALL(hipGetDeviceCount(&device_count));
  logFile << "\t" << device_count << " devices available!\n";

  // creating the subgraphs
  logFile << "Creating the subgraphs ...\n";
  std::vector<Type1SubGraph> type1_subgraphs(device_count);
  for (int device_idx = 0; device_idx < device_count; device_idx++)
    type1_subgraphs[device_idx].init(g, device_count, device_idx);
  logFile << "All the Type1SubGraph created!\n";

  // map
  logFile << "Start mapping ...\n";
  int normal_vertex_number_each_partition = (super_graph_vertex_num - 1) / device_count + 1;
  for (int device_idx = 0; device_idx < device_count; device_idx++)
  {
    logFile << "\tMapping part " << device_idx << " ...\n";
    vidType partition_start_vertex = normal_vertex_number_each_partition * device_idx;
    vidType partition_end_vertex = (partition_start_vertex + normal_vertex_number_each_partition > super_graph_vertex_num) ? super_graph_vertex_num : partition_start_vertex + normal_vertex_number_each_partition;
    for (vidType u = partition_start_vertex; u < partition_end_vertex; u++)
    {
      vidType u_deg = g.out_degree(u);
      for (vidType v_idx = 0; v_idx < u_deg; v_idx++)
      {
        vidType v = g.N(u, v_idx);
        int v_partition_idx = v % normal_vertex_number_each_partition;
      }
    }
  }

  // end and exit
  logFile << "Destoying Type1SubGraphs ...\n";
  for (int device_idx = 0; device_idx < device_count; device_idx++)
    type1_subgraphs[device_idx].destroy();
  logFile << "All the Type1SubGraphs have been destoyed!\n";
  logFile << "Gracefully finishing ...\n";
  logFile.close();
}

