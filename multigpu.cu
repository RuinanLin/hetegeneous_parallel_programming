#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
//#include "edgelist.h"
#include "graph_gpu.h"
#include "graph_partition.h"
#include "scheduler.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"

typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;
#include "bs_warp_edge.cuh"
#include "bs_warp_vertex.cuh"
#include <thread>

#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <algorithm>
#include <tuple>

typedef std::pair<vidType, vidType> directedEdge;

int get_type3_subgraph_num(int device_count);
void get_type3_subgraph_tuple(int *this_subgraph_tuple, int this_type3_subgraph_idx, int num_partitions);
int get_type3_subgraph_idx(int device_count, vidType u, vidType v, vidType w);
void sort_three_vertices(vidType *u, vidType *v, vidType *w);

/**************************************** Definition of Type1SubGraph ***************************************/

class Type1SubGraph {
  protected:
    vidType super_num_vertices;           // how many vertices in the super graph
    int num_partitions;                   // the super graph is partitioned into how many partitions
    int this_partition_idx;               // the index of the partition corresponding to this subgraph
    vidType start_vertex_idx;             // the start index of the vertices in this partition, in the global view
    vidType this_num_vertices;            // how many vertices in this partition
    std::ofstream logFile;                // its own logFile

    vidType *edges;                       // column indices of CSR format, starting point only inner
    eidType *row_pointers;                // row pointers of CSR format, starting point only inner

    std::vector<directedEdge> temp_edges; // for the process of generating, allowing start from outer
    int creation_finished;                // whether the subgraph has finished creating

  public:
    void init(Graph &g, int num_partitions, int this_partition_idx);
    void destroy();
    void add_edge(vidType from, vidType to);
    void reduce();
    vidType get_out_degree(vidType u);
    vidType N(vidType u, vidType n);
};

// initialize the subgraph
void Type1SubGraph::init(Graph &g, int num_part, int this_part_idx)
{
  // initialize its own logFile
  std::string file_name = "Type1SubGraph_log" + std::to_string(this_part_idx) + ".txt";
  logFile.open(file_name);
  if (!logFile)
  {
    std::cerr << "Cannot open " << file_name << "\n";
    exit(-1);
  }
  logFile << "logFile created!\nType1SubGraph " << this_part_idx << " initialization starts ...\n";

  // initialize private variables
  super_num_vertices = g.V();
  num_partitions = num_part;
  this_partition_idx = this_part_idx;
  vidType normal_vertex_number_each_partition = (super_num_vertices - 1) / num_partitions + 1;
  start_vertex_idx = normal_vertex_number_each_partition * this_partition_idx;
  this_num_vertices = (this_partition_idx == num_partitions - 1) ? (super_num_vertices - normal_vertex_number_each_partition * (num_partitions - 1)) : normal_vertex_number_each_partition;
  creation_finished = 0;
  logFile << "\tPrivate variables initialized!\n";
  logFile << "\t\tsuper_num_vertices = " << super_num_vertices << "\n";
  logFile << "\t\tnum_partitions = " << num_partitions << "\n";
  logFile << "\t\tthis_partition_idx = " << this_partition_idx << "\n";
  logFile << "\t\tstart_vertex_idx = " << start_vertex_idx << "\n";
  logFile << "\t\tthis_num_vertices = " << this_num_vertices << "\n";

  // initialize the CSR
  logFile << "\tAllocate row_pointers ...\n";
  edges = NULL;   // no edges now
  row_pointers = (eidType *)malloc((this_num_vertices + 1) * sizeof(eidType));
  for (eidType u = 0; u < this_num_vertices + 1; u++)
    row_pointers[u] = 0;

  // finish initialization and exit
  logFile << "Initialization succeeded!\n";
}

// destroy the subgraph
void Type1SubGraph::destroy()
{
  logFile << "Start destroying Type1SubGraph " << this_partition_idx << " ...\n";

  // free the "edges_from_outside" and "edges_to_outside"
  logFile << "\tFree the allocated memory ...\n";
  free(edges);
  free(row_pointers);
  logFile << "\tAllocated memory freed!\n";

  // close the logFile
  logFile << "Gracefully finishing ...\n";
  logFile.close();
}

// push a new edge to the temp_edges
void Type1SubGraph::add_edge(vidType from, vidType to)
{
  // refresh the counter of 'from'
  row_pointers[from - start_vertex_idx]++;

  // create a pair
  directedEdge edge(from, to);
  temp_edges.push_back(edge);
}

// reorder the temp_edges and create the final CSR format
void Type1SubGraph::reduce()
{
  // allocate memory for 'edges'
  edges = (vidType *)malloc(temp_edges.size() * sizeof(vidType));

  // perform scanning on 'row_pointers', thus getting the indices
  for (vidType counter_idx = 0; counter_idx < this_num_vertices; counter_idx++)
    row_pointers[counter_idx + 1] += row_pointers[counter_idx];

  // pop the edges from back one by one and place it to the correct place
  while (temp_edges.size() > 0)
  {
    // pop the last pair out
    directedEdge edge = temp_edges[temp_edges.size() - 1];
    temp_edges.pop_back();

    // put it into the right place in 'edges'
    edges[--row_pointers[edge.first - start_vertex_idx]] = edge.second;
  }

  // pull up the 'creation_finished' flag
  creation_finished = 1;

  // sort and print the result into the logFile
  logFile << "Type1SubGraph " << this_partition_idx << " has finished reducing!\n";
  for (vidType u = start_vertex_idx; u < start_vertex_idx + this_num_vertices; u++)
  {
    std::sort(edges + row_pointers[u - start_vertex_idx], edges + row_pointers[u + 1 - start_vertex_idx]);
    logFile << "\t" << u << ": ";
    vidType u_deg = get_out_degree(u);
    for (vidType v_idx = 0; v_idx < u_deg; v_idx++)
      logFile << N(u, v_idx) << " ";
    logFile << "\n";
  }
}

// get the out-degree of vertex u
vidType Type1SubGraph::get_out_degree(vidType u)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'get_out_degree()' is called!\n";
    exit(-1);
  }
  return row_pointers[u - start_vertex_idx + 1] - row_pointers[u - start_vertex_idx];
}

// get the n-th neighbor of u
vidType Type1SubGraph::N(vidType u, vidType n)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'N()' is called!\n";
    exit(-1);
  }
  return edges[row_pointers[u - start_vertex_idx] + n];
}

/**************************************** Definition of Type3SubGraph ***************************************/

class Type3SubGraph {
  protected:
    int this_type3_subgraph_idx;                                            // the global index among all the Type3SubGraphs
    int num_partitions;                                                     // how many partitions in the super graph
    int this_subgraph_tuple[3];                                             // records the three indices of the partitions, sorted
    vidType super_num_vertices;                                             // how many vertices in the super graph
    vidType partition_num_vertices_tuple[3];                                // how many vertices are there in each partition
    vidType partition_start_vertex_idx_tuple[3]; // each partition starts at which vertex
    std::ofstream logFile;                                                  // its own logFile

    vidType *edges_0_to_1;
    eidType *row_pointers_0_to_1;
    std::vector<directedEdge> temp_edges_0_to_1;

    vidType *edges_0_to_2;
    eidType *row_pointers_0_to_2;
    std::vector<directedEdge> temp_edges_0_to_2;

    vidType *edges_1_to_2;
    eidType *row_pointers_1_to_2;
    std::vector<directedEdge> temp_edges_1_to_2;

    int creation_finished;

  public:
    void init(Graph &g, int num_part, int this_type3_subgraph_index);
    void destroy();
    void add_edge(vidType u, int u_partition_idx, vidType v, int v_partition_idx);
    void reduce();
    vidType get_out_degree_0_to_1(vidType u);
    vidType get_out_degree_0_to_2(vidType u);
    vidType get_out_degree_1_to_2(vidType v);
    vidType N_0_to_1(vidType u, vidType n);
    vidType N_0_to_2(vidType u, vidType n);
    vidType N_1_to_2(vidType v, vidType n);
};

// initialize the subgraph
void Type3SubGraph::init(Graph &g, int num_part, int this_type3_subgraph_index)
{
  // initialize its own logFile
  std::string file_name = "Type3SubGraph_log" + std::to_string(this_type3_subgraph_index) + ".txt";
  logFile.open(file_name);
  if (!logFile)
  {
    std::cerr << "Cannot open " << file_name << "\n";
    exit(-1);
  }
  logFile << "logFile created!\nType3SubGraph " << this_type3_subgraph_index << " initialization starts ...\n";

  // initialize private variables
  this_type3_subgraph_idx = this_type3_subgraph_index;
  num_partitions = num_part;
  get_type3_subgraph_tuple(this_subgraph_tuple, this_type3_subgraph_idx, num_partitions);
  super_num_vertices = g.V();
  vidType normal_vertex_number_each_partition = (super_num_vertices - 1) / num_partitions + 1;
  partition_num_vertices_tuple[0] = normal_vertex_number_each_partition;
  partition_num_vertices_tuple[1] = normal_vertex_number_each_partition;
  partition_num_vertices_tuple[2] = (this_subgraph_tuple[2] == num_partitions - 1) ? (super_num_vertices - normal_vertex_number_each_partition * (num_partitions - 1)) : normal_vertex_number_each_partition;
  partition_start_vertex_idx_tuple[0] = normal_vertex_number_each_partition * this_subgraph_tuple[0];
  partition_start_vertex_idx_tuple[1] = normal_vertex_number_each_partition * this_subgraph_tuple[1];
  partition_start_vertex_idx_tuple[2] = normal_vertex_number_each_partition * this_subgraph_tuple[2];
  creation_finished = 0;
  logFile << "\tPrivate variables initialized!\n";
  logFile << "\t\tthis_type3_subgraph_idx = " << this_type3_subgraph_idx << "\n";
  logFile << "\t\tnum_partitions = " << num_partitions << "\n";
  logFile << "\t\tthis_subgraph_tuple = (" << this_subgraph_tuple[0] << ", " << this_subgraph_tuple[1] << ", " << this_subgraph_tuple[2] << ")\n";
  logFile << "\t\tsuper_num_vertices = " << super_num_vertices << "\n";
  logFile << "\t\tpartition_num_vertices_tuple = (" << partition_num_vertices_tuple[0] << ", " << partition_num_vertices_tuple[1] << ", " << partition_num_vertices_tuple[2] << ")\n";
  logFile << "\t\tpartition_start_vertex_idx_tuple = (" << partition_start_vertex_idx_tuple[0] << ", " << partition_start_vertex_idx_tuple[1] << ", " << partition_start_vertex_idx_tuple[2] << ")\n";

  // allocate memory
  logFile << "\tAllocate row_pointers ...\n";
  row_pointers_0_to_1 = (eidType *)malloc((partition_num_vertices_tuple[0] + 1) * sizeof(eidType));
  for (eidType u = 0; u < partition_num_vertices_tuple[0] + 1; u++)
    row_pointers_0_to_1[u] = 0;
  row_pointers_0_to_2 = (eidType *)malloc((partition_num_vertices_tuple[0] + 1) * sizeof(eidType));
  for (eidType u = 0; u < partition_num_vertices_tuple[0] + 1; u++)
    row_pointers_0_to_2[u] = 0;
  row_pointers_1_to_2 = (eidType *)malloc((partition_num_vertices_tuple[1] + 1) * sizeof(eidType));
  for (eidType u = 0; u < partition_num_vertices_tuple[1] + 1; u++)
    row_pointers_1_to_2[u] = 0;
  
  // finish initialization and exit
  logFile << "Initialization finished!\n";
}

// destroy the subgraph
void Type3SubGraph::destroy()
{
  logFile << "Start destroying Type3SubGraph " << this_type3_subgraph_idx << " ...\n";

  // free the allocated memory
  logFile << "\tFree the allocated memory ...\n";
  free(edges_0_to_1);
  free(row_pointers_0_to_1);
  free(edges_0_to_2);
  free(row_pointers_0_to_2);
  free(edges_1_to_2);
  free(row_pointers_1_to_2);
  logFile << "\tAllocated memory freed!\n";

  // close the logFile
  logFile << "Gracefully finishing ...\n";
  logFile.close();
}

// add (u, v) to the Type3SubGraph (u < v is guaranteed)
void Type3SubGraph::add_edge(vidType u, int u_partition_idx, vidType v, int v_partition_idx)
{
  directedEdge edge(u, v);

  // there are 3 cases if u < v is guaranteed
  if (u_partition_idx == this_subgraph_tuple[0] && v_partition_idx == this_subgraph_tuple[1])
  {
    temp_edges_0_to_1.push_back(edge);
    row_pointers_0_to_1[u - partition_start_vertex_idx_tuple[0]]++;
  }
  else if (u_partition_idx == this_subgraph_tuple[0] && v_partition_idx == this_subgraph_tuple[2])
  {
    temp_edges_0_to_2.push_back(edge);
    row_pointers_0_to_2[u - partition_start_vertex_idx_tuple[0]]++;
  }
  else
  {
    temp_edges_1_to_2.push_back(edge);
    row_pointers_1_to_2[u - partition_start_vertex_idx_tuple[1]]++;
  }
}

// reorder and create the CSR format
void Type3SubGraph::reduce()
{
  // allocate memory for 'edges'
  edges_0_to_1 = (vidType *)malloc(temp_edges_0_to_1.size() * sizeof(vidType));
  edges_0_to_2 = (vidType *)malloc(temp_edges_0_to_2.size() * sizeof(vidType));
  edges_1_to_2 = (vidType *)malloc(temp_edges_1_to_2.size() * sizeof(vidType));

  // perform scanning on row_pointers
  for (vidType counter_idx = 0; counter_idx < partition_num_vertices_tuple[0]; counter_idx++)
  {
    row_pointers_0_to_1[counter_idx + 1] += row_pointers_0_to_1[counter_idx];
    row_pointers_0_to_2[counter_idx + 1] += row_pointers_0_to_2[counter_idx];
  }
  for (vidType counter_idx = 0; counter_idx < partition_num_vertices_tuple[1]; counter_idx++)
    row_pointers_1_to_2[counter_idx + 1] += row_pointers_1_to_2[counter_idx];

  // pop the edges from back one by one and place it to the correct place
  vidType start_idx = partition_start_vertex_idx_tuple[0];
  while (temp_edges_0_to_1.size() > 0)
  {
    directedEdge edge = temp_edges_0_to_1[temp_edges_0_to_1.size() - 1];
    edges_0_to_1[--row_pointers_0_to_1[edge.first - start_idx]] = edge.second;
    temp_edges_0_to_1.pop_back();
  }
  while (temp_edges_0_to_2.size() > 0)
  {
    directedEdge edge = temp_edges_0_to_2[temp_edges_0_to_2.size() - 1];
    edges_0_to_2[--row_pointers_0_to_2[edge.first - start_idx]] = edge.second;
    temp_edges_0_to_2.pop_back();
  }
  start_idx = partition_start_vertex_idx_tuple[1];
  while (temp_edges_1_to_2.size() > 0)
  {
    directedEdge edge = temp_edges_1_to_2[temp_edges_1_to_2.size() - 1];
    edges_1_to_2[--row_pointers_1_to_2[edge.first - start_idx]] = edge.second;
    temp_edges_1_to_2.pop_back();
  }

  // pull up the 'creation_finished' flag
  creation_finished = 1;

  // sort and print the result into the logFile
  logFile << "Type3SubGraph " << this_type3_subgraph_idx << " has finished reducing!\n";

  logFile << "\t" << this_subgraph_tuple[0] << " to " << this_subgraph_tuple[1] << ":\n";
  vidType partition_start_vertex_idx_0 = partition_start_vertex_idx_tuple[0];
  vidType partition_num_vertices_0 = partition_num_vertices_tuple[0];
  for (vidType u = partition_start_vertex_idx_0; u < partition_start_vertex_idx_0 + partition_num_vertices_0; u++)
  {
    std::sort(edges_0_to_1 + row_pointers_0_to_1[u - partition_start_vertex_idx_0], edges_0_to_1 + row_pointers_0_to_1[u + 1 - partition_start_vertex_idx_0]);
    logFile << "\t\t" << u << ": ";
    vidType u_0_to_1_deg = get_out_degree_0_to_1(u);
    for (vidType v_idx = 0; v_idx < u_0_to_1_deg; v_idx++)
      logFile << N_0_to_1(u, v_idx) << " ";
    logFile << "\n";
  }

  logFile << "\t" << this_subgraph_tuple[0] << " to " << this_subgraph_tuple[2] << ":\n";
  for (vidType u = partition_start_vertex_idx_0; u < partition_start_vertex_idx_0 + partition_num_vertices_0; u++)
  {
    std::sort(edges_0_to_2 + row_pointers_0_to_2[u - partition_start_vertex_idx_0], edges_0_to_2 + row_pointers_0_to_2[u + 1 - partition_start_vertex_idx_0]);
    logFile << "\t\t" << u << ": ";
    vidType u_0_to_2_deg = get_out_degree_0_to_2(u);
    for (vidType w_idx = 0; w_idx < u_0_to_2_deg; w_idx++)
      logFile << N_0_to_2(u, w_idx) << " ";
    logFile << "\n";
  }

  logFile << "\t" << this_subgraph_tuple[1] << " to " << this_subgraph_tuple[2] << ":\n";
  vidType partition_start_vertex_idx_1 = partition_start_vertex_idx_tuple[1];
  vidType partition_num_vertices_1 = partition_num_vertices_tuple[1];
  for (vidType v = partition_start_vertex_idx_1; v < partition_start_vertex_idx_1 + partition_num_vertices_1; v++)
  {
    std::sort(edges_1_to_2 + row_pointers_1_to_2[v - partition_start_vertex_idx_1], edges_1_to_2 + row_pointers_1_to_2[v + 1 - partition_start_vertex_idx_1]);
    logFile << "\t\t" << v << ": ";
    vidType v_1_to_2_deg = get_out_degree_1_to_2(v);
    for (vidType w_idx = 0; w_idx < v_1_to_2_deg; w_idx++)
      logFile << N_1_to_2(v, w_idx) << " ";
    logFile << "\n";
  }
}

vidType Type3SubGraph::get_out_degree_0_to_1(vidType u)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'get_out_degree_0_to_1()' is called!\n";
    exit(-1);
  }
  vidType start_vertex_idx = partition_start_vertex_idx_tuple[0];
  return row_pointers_0_to_1[u - start_vertex_idx + 1] - row_pointers_0_to_1[u - start_vertex_idx];
}

vidType Type3SubGraph::get_out_degree_0_to_2(vidType u)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'get_out_degree_0_to_2()' is called!\n";
    exit(-1);
  }
  vidType start_vertex_idx = partition_start_vertex_idx_tuple[0];
  return row_pointers_0_to_2[u - start_vertex_idx + 1] - row_pointers_0_to_2[u - start_vertex_idx];
}

vidType Type3SubGraph::get_out_degree_1_to_2(vidType v)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'get_out_degree_1_to_2()' is called!\n";
    exit(-1);
  }
  vidType start_vertex_idx = partition_start_vertex_idx_tuple[1];
  return row_pointers_1_to_2[v - start_vertex_idx + 1] - row_pointers_1_to_2[v - start_vertex_idx];
}

vidType Type3SubGraph::N_0_to_1(vidType u, vidType n)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'N_0_to_1()' is called!\n";
    exit(-1);
  }
  vidType start_vertex_idx = partition_start_vertex_idx_tuple[0];
  return edges_0_to_1[row_pointers_0_to_1[u - start_vertex_idx] + n];
}

vidType Type3SubGraph::N_0_to_2(vidType u, vidType n)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'N_0_to_2()' is called!\n";
    exit(-1);
  }
  vidType start_vertex_idx = partition_start_vertex_idx_tuple[0];
  return edges_0_to_2[row_pointers_0_to_2[u - start_vertex_idx] + n];
}

vidType Type3SubGraph::N_1_to_2(vidType v, vidType n)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'N_1_to_2()' is called!\n";
    exit(-1);
  }
  vidType start_vertex_idx = partition_start_vertex_idx_tuple[1];
  return edges_1_to_2[row_pointers_1_to_2[v - start_vertex_idx] + n];
}

/**************************************** Definition of tool functions ***************************************/

// given the device_count, calculate how many Type3SubGraphs
int get_type3_subgraph_num(int device_count)
{
  return (device_count * (device_count - 1) * (device_count - 2)) / 6;
}

// given the global index of Type3SubGraph, calculate its 3-tuple
void get_type3_subgraph_tuple(int *this_subgraph_tuple, int this_type3_subgraph_idx, int num_partitions)
{
  // check whether the requirement is legal, in the legal range
  int type3_subgraph_num = get_type3_subgraph_num(num_partitions);
  if (this_type3_subgraph_idx < 0 || this_type3_subgraph_idx >= type3_subgraph_num)
  {
    std::cout << "Error! 'this_type3_subgraph_idx' out of range in 'get_type3_subgraph_tuple()' call!\n";
    exit(-1);
  }

  // travers and attempt
  int first;
  int second;
  int third;
  for (first = 0; first < num_partitions - 2; first++)
  {
    int num_subgraphs_with_first = (num_partitions - first - 1) * (num_partitions - first - 2) / 2;
    if (num_subgraphs_with_first <= this_type3_subgraph_idx)
      this_type3_subgraph_idx -= num_subgraphs_with_first;
    else
    {
      this_subgraph_tuple[0] = first;
      for (second = first + 1; second < num_partitions - 1; second++)
      {
        int num_subgraphs_with_second = num_partitions - second - 1;
        if (num_subgraphs_with_second <= this_type3_subgraph_idx)
          this_type3_subgraph_idx -= num_subgraphs_with_second;
        else
        {
          this_subgraph_tuple[1] = second;
          third = second + 1 + this_type3_subgraph_idx;
          this_subgraph_tuple[2] = third;
          break;
        }
      }
      break;
    }
  }
}

// given the device_count and three seperate partition index, calculate the global index of the Type3SubGraph
int get_type3_subgraph_idx(int device_count, int u_device_idx, int v_device_idx, int w_device_idx)
{
  // sort the three partitions into increasing order
  sort_three_vertices(&u_device_idx, &v_device_idx, &w_device_idx);

  // accumulate index
  int type3_subgraph_idx = 0;
  for (vidType ui = 0; ui < u_device_idx; ui++)
    type3_subgraph_idx += (device_count - ui - 1) * (device_count - ui - 2) / 2;
  for (vidType vi = u_device_idx + 1; vi < v_device_idx; vi++)
    type3_subgraph_idx += device_count - vi - 1;
  type3_subgraph_idx += w_device_idx - v_device_idx - 1;
  return type3_subgraph_idx;
}

// sort the three vertices into increasing order
void sort_three_vertices(vidType *u, vidType *v, vidType *w)
{
  if (*u > *v)
  {
    vidType temp = *v;
    *v = *u;
    *u = temp;
  }
  if (*v > *w)
  {
    vidType temp = *w;
    *w = *v;
    *v = temp;
  }
  if (*u > *v)
  {
    vidType temp = *v;
    *v = *u;
    *u = temp;
  }
}

/**************************************** Definition of TCSolver ***************************************/

void TCSolver(Graph &g, uint64_t &total, int n_gpus, int chunk_size) {
  // start TCSolver
  std::string file_name = "log.txt";
  std::ofstream logFile(file_name);
  if (!logFile)
  {
    std::cerr << "Cannot open " << file_name << "\n";
    exit(-1);
  }
  logFile << "TCSolver starts ...\n";

  // read important information out from the super graph
  logFile << "Reading input graph ...\n";
  vidType super_graph_vertex_num = g.V();
  logFile << "|V| = " << super_graph_vertex_num << "\n";

  // get the device_count of the system
  logFile << "Looking for devices ...\n";
  int device_count;
  CUDA_SAFE_CALL(hipGetDeviceCount(&device_count));
  logFile << "\t" << device_count << " devices available!\n";

  // creating the subgraphs
  logFile << "Creating the subgraphs ...\n";
  std::vector<Type1SubGraph> type1_subgraphs(device_count);
  for (int device_idx = 0; device_idx < device_count; device_idx++)
    type1_subgraphs[device_idx].init(g, device_count, device_idx);
  logFile << "\tAll the Type1SubGraphs created!\n";
  int type3_subgraph_num = get_type3_subgraph_num(device_count);
  std::vector<Type3SubGraph> type3_subgraphs(device_count);
  for (int type3_subgraph_idx = 0; type3_subgraph_idx < type3_subgraph_num; type3_subgraph_idx++)
    type3_subgraphs[type3_subgraph_idx].init(g, device_count, type3_subgraph_idx);
  logFile << "\tAll the Type3SubGraphs created!\n";

  // map
  logFile << "Start mapping ...\n";
  int normal_vertex_number_each_partition = (super_graph_vertex_num - 1) / device_count + 1;
  for (int device_idx = 0; device_idx < device_count; device_idx++)
  {
    logFile << "\tMapping part " << device_idx << " ...\n";
    vidType partition_start_vertex = normal_vertex_number_each_partition * device_idx;
    vidType partition_end_vertex = (partition_start_vertex + normal_vertex_number_each_partition > super_graph_vertex_num) ? super_graph_vertex_num : partition_start_vertex + normal_vertex_number_each_partition;
    for (vidType u = partition_start_vertex; u < partition_end_vertex; u++)
    {
      vidType u_deg = g.out_degree(u);
      for (vidType v_idx = 0; v_idx < u_deg; v_idx++)
      {
        vidType v = g.N(u, v_idx);
        int v_partition_idx = v / normal_vertex_number_each_partition;
        if (device_idx == v_partition_idx)  // inner edge
          type1_subgraphs[device_idx].add_edge(u, v);
        else  // cross edge
        {
          // First, record in their coresponding Type1SubGraphs
          type1_subgraphs[device_idx].add_edge(u, v);
          type1_subgraphs[v_partition_idx].add_edge(v, u);

          // Second, record into all the related Type3SubGraphs
          for (vidType w_device_idx = 0; w_device_idx < device_count; w_device_idx++)
          {
            if (w_device_idx != device_idx && w_device_idx != v_partition_idx)
            {
              int type3_subgraph_idx = get_type3_subgraph_idx(device_count, device_idx, v_partition_idx, w_device_idx);
              if (u < v)
              {
                type3_subgraphs[type3_subgraph_idx].add_edge(u, device_idx, v, v_partition_idx);
              }
              else
                type3_subgraphs[type3_subgraph_idx].add_edge(v, v_partition_idx, u, device_idx);
            }
          }
        }
      }
    }
  }
  logFile << "Finish mapping!\n";

  // reduce
  logFile << "Start reducing ...\n";
  for (int device_idx = 0; device_idx < device_count; device_idx++)
    type1_subgraphs[device_idx].reduce();
  for (int type3_subgraph_idx = 0; type3_subgraph_idx < type3_subgraph_num; type3_subgraph_idx++)
    type3_subgraphs[type3_subgraph_idx].reduce();
  logFile << "Finish reducing ...\n";

  // end and exit
  logFile << "Destoying Type1SubGraphs ...\n";
  for (int device_idx = 0; device_idx < device_count; device_idx++)
    type1_subgraphs[device_idx].destroy();
  logFile << "All the Type1SubGraphs have been destoyed!\n";
  logFile << "Destoying Type3SubGraphs ...\n";
  for (int type3_subgraph_idx = 0; type3_subgraph_idx < type3_subgraph_num; type3_subgraph_idx++)
    type3_subgraphs[type3_subgraph_idx].destroy();
  logFile << "All the Type3SubGraphs have been destoyed!\n";
  logFile << "Gracefully finishing ...\n";
  logFile.close();
}
