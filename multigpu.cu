#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
//#include "edgelist.h"
#include "graph_gpu.h"
#include "graph_partition.h"
#include "scheduler.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"

typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;
#include "bs_warp_edge.cuh"
#include "bs_warp_vertex.cuh"
#include <thread>

#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <algorithm>
#include <tuple>

typedef std::pair<vidType, vidType> directedEdge;
typedef std::tuple<int, int, int> type3Tuple;

int get_type3_subgraph_num(int device_count);
type3Tuple get_type3_subgraph_tuple(int this_type3_subgraph_idx, int num_partitions);
int get_type3_subgraph_idx(int device_count, vidType u, vidType v, vidType w);
void sort_three_vertices(vidType *u, vidType *v, vidType *w);

/**************************************** Definition of Type1SubGraph ***************************************/

class Type1SubGraph {
  protected:
    vidType super_num_vertices;           // how many vertices in the super graph
    int num_partitions;                   // the super graph is partitioned into how many partitions
    int this_partition_idx;               // the index of the partition corresponding to this subgraph
    vidType start_vertex_idx;             // the start index of the vertices in this partition, in the global view
    vidType this_num_vertices;            // how many vertices in this partition
    std::ofstream logFile;                // its own logFile

    vidType *edges;                       // column indices of CSR format, starting point only inner
    eidType *row_pointers;                // row pointers of CSR format, starting point only inner

    std::vector<directedEdge> temp_edges; // for the process of generating, allowing start from outer
    int creation_finished;                // whether the subgraph has finished creating

  public:
    void init(Graph &g, int num_partitions, int this_partition_idx);
    void destroy();
    void add_edge(vidType from, vidType to);
    void reduce();
    vidType get_out_degree(vidType u);
    vidType N(vidType u, vidType n);
};

// initialize the subgraph
void Type1SubGraph::init(Graph &g, int num_part, int this_part_idx)
{
  // initialize its own logFile
  std::string file_name = "Type1SubGraph_log" + std::to_string(this_part_idx) + ".txt";
  logFile.open(file_name);
  if (!logFile)
  {
    std::cerr << "Cannot open " << file_name << "\n";
    exit(-1);
  }
  logFile << "logFile created!\nType1SubGraph " << this_part_idx << " initialization starts ...\n";

  // initialize private variables
  super_num_vertices = g.V();
  num_partitions = num_part;
  this_partition_idx = this_part_idx;
  vidType normal_vertex_number_each_partition = (super_num_vertices - 1) / num_partitions + 1;
  start_vertex_idx = normal_vertex_number_each_partition * this_partition_idx;
  this_num_vertices = (this_partition_idx == num_partitions - 1) ? (super_num_vertices - normal_vertex_number_each_partition * (num_partitions - 1)) : normal_vertex_number_each_partition;
  creation_finished = 0;
  logFile << "\tPrivate variables initialized!\n";
  logFile << "\t\tsuper_num_vertices = " << super_num_vertices << "\n";
  logFile << "\t\tnum_partitions = " << num_partitions << "\n";
  logFile << "\t\tthis_partition_idx = " << this_partition_idx << "\n";
  logFile << "\t\tstart_vertex_idx = " << start_vertex_idx << "\n";
  logFile << "\t\tthis_num_vertices = " << this_num_vertices << "\n";

  // initialize the CSR
  logFile << "\tAllocate row_pointers ...\n";
  edges = NULL;   // no edges now
  row_pointers = (eidType *)malloc((this_num_vertices + 1) * sizeof(eidType));
  for (eidType u = 0; u < this_num_vertices + 1; u++)
    row_pointers[u] = 0;

  // finish initialization and exit
  logFile << "Initialization succeeded!\n";
}

// destroy the subgraph
void Type1SubGraph::destroy()
{
  logFile << "Start destroying Type1SubGraph " << this_partition_idx << " ...\n";

  // free the "edges_from_outside" and "edges_to_outside"
  logFile << "\tFree the allocated memory ...\n";
  free(edges);
  free(row_pointers);
  logFile << "\tAllocated memory freed!\n";

  // close the logFile
  logFile << "Gracefully finishing ...\n";
  logFile.close();
}

// push a new edge to the temp_edges
void Type1SubGraph::add_edge(vidType from, vidType to)
{
  // refresh the counter of 'from'
  row_pointers[from - start_vertex_idx]++;

  // create a pair
  directedEdge edge(from, to);
  temp_edges.push_back(edge);
}

// reorder the temp_edges and create the final CSR format
void Type1SubGraph::reduce()
{
  // allocate memory for 'edges'
  edges = (vidType *)malloc(temp_edges.size() * sizeof(vidType));

  // perform scanning on 'row_pointers', thus getting the indices
  for (vidType counter_idx = 0; counter_idx < this_num_vertices; counter_idx++)
    row_pointers[counter_idx + 1] += row_pointers[counter_idx];

  // pop the edges from back one by one and place it to the correct place
  while (temp_edges.size() > 0)
  {
    // pop the last pair out
    directedEdge edge = temp_edges[temp_edges.size() - 1];
    temp_edges.pop_back();

    // put it into the right place in 'edges'
    edges[--row_pointers[edge.first - start_vertex_idx]] = edge.second;
  }

  // pull up the 'creation_finished' flag
  creation_finished = 1;

  // sort and print the result into the logFile
  logFile << "Type1SubGraph " << this_partition_idx << " has finished reducing!\n";
  for (vidType u = start_vertex_idx; u < start_vertex_idx + this_num_vertices; u++)
  {
    std::sort(edges + row_pointers[u - start_vertex_idx], edges + row_pointers[u + 1 - start_vertex_idx]);
    logFile << "\t" << u << ": ";
    vidType u_deg = get_out_degree(u);
    for (vidType v_idx = 0; v_idx < u_deg; v_idx++)
      logFile << N(u, v_idx) << " ";
    logFile << "\n";
  }
}

// get the out-degree of vertex u
vidType Type1SubGraph::get_out_degree(vidType u)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'get_out_degree()' is called!\n";
    exit(-1);
  }
  return row_pointers[u - start_vertex_idx + 1] - row_pointers[u - start_vertex_idx];
}

// get the n-th neighbor of u
vidType Type1SubGraph::N(vidType u, vidType n)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'N()' is called!\n";
    exit(-1);
  }
  return edges[row_pointers[u - start_vertex_idx] + n];
}

/**************************************** Definition of Type3SubGraph ***************************************/

class Type3SubGraph {
  protected:
    int this_type3_subgraph_idx;                                            // the global index among all the Type3SubGraphs
    int num_partitions;                                                     // how many partitions in the super graph
    type3Tuple this_subgraph_tuple;                                         // records the three indices of the partitions, sorted
    vidType super_num_vertices;                                             // how many vertices in the super graph
    std::tuple<vidType, vidType, vidType> partition_num_vertices_tuple;     // how many vertices are there in each partition
    std::tuple<vidType, vidType, vidType> partition_start_vertex_idx_tuple; // each partition starts at which vertex
    std::ofstream logFile;                                                  // its own logFile

    vidType *edges_0_to_1;
    eidType *row_pointers_0_to_1;
    std::vector<directedEdge> temp_edges_0_to_1;

    vidType *edges_0_to_2;
    eidType *row_pointers_0_to_2;
    std::vector<directedEdge> temp_edges_0_to_2;

    vidType *edges_1_to_2;
    eidType *row_pointers_1_to_2;
    std::vector<directedEdge> temp_edges_1_to_2;

    int creation_finished;

  public:
    void init(Graph &g, int num_part, int this_type3_subgraph_index);
    void destroy();
    void add_edge(vidType u, int u_partition_idx, vidType v, int v_partition_idx);
    void recude();
    vidType get_out_degree_0_to_1(vidType u);
    vidType get_out_degree_0_to_2(vidType u);
    vidType get_out_degree_1_to_2(vidType v);
    vidType N_0_to_1(vidType u, vidType n);
    vidType N_0_to_2(vidType u, vidType n);
    vidType N_1_to_2(vidType v, vidType n);
}

// initialize the subgraph
void Type3SubGraph::init(Graph &g, int num_part, int this_type3_subgraph_index)
{
  // initialize its own logFile
  std::string file_name = "Type3SubGraph_log" + std::to_string(this_type3_subgraph_index) + ".txt";
  logFile.open(file_name);
  if (!logFile)
  {
    std::cerr << "Cannot open " << file_name << "\n";
    exit(-1);
  }
  logFile << "logFile created!\nType3SubGraph " << this_type3_subgraph_index << " initialization starts ...\n";

  // initialize private variables
  this_type3_subgraph_idx = this_type3_subgraph_index;
  num_partitions = num_part;
  this_subgraph_tuple = get_type3_subgraph_tuple(this_type3_subgraph_idx, num_partitions);
  super_num_vertices = g.V();
  vidType normal_vertex_number_each_partition = (super_num_vertices - 1) / num_partitions + 1;
  std::get<0>(partition_num_vertices_tuple) = normal_vertex_number_each_partition;
  std::get<1>(partition_num_vertices_tuple) = normal_vertex_number_each_partition;
  std::get<2>(partition_num_vertices_tuple) = (std::get<2>(this_subgraph_tuple) == num_partitions - 1) ? (super_num_vertices - normal_vertex_number_each_partition * (num_partitions - 1)) : normal_vertex_number_each_partition;
  std::get<0>(partition_start_vertex_idx_tuple) = normal_vertex_number_each_partition * std::get<0>(this_subgraph_tuple);
  std::get<1>(partition_start_vertex_idx_tuple) = normal_vertex_number_each_partition * std::get<1>(this_subgraph_tuple);
  std::get<2>(partition_start_vertex_idx_tuple) = normal_vertex_number_each_partition * std::get<2>(this_subgraph_tuple);
  creation_finished = 0;
  logFile << "\tPrivate variables initialized!\n";
  logFile << "\t\tthis_type3_subgraph_idx = " << this_type3_subgraph_idx << "\n";
  logFile << "\t\tnum_partitions = " << num_partitions << "\n";
  logFile << "\t\tthis_subgraph_tuple = (" << std::get<0>(this_subgraph_tuple) << ", " << std::get<1>(this_subgraph_tuple) << ", " << std::get<2>(this_subgraph_tuple) << ")\n";
  logFile << "\t\tsuper_num_vertices = " << super_num_vertices << "\n";
  logFile << "\t\tpartition_num_vertices_tuple = (" << std::get<0>(partition_num_vertices_tuple) << ", " << std::get<1>(partition_num_vertices_tuple) << ", " << std::get<2>(partition_num_vertices_tuple) << ")\n";
  logFile << "\t\tpartition_start_vertex_idx_tuple = (" << std::get<0>(partition_start_vertex_idx_tuple) << ", " << std::get<1>(partition_start_vertex_idx_tuple) << ", " << std::get<2>(partition_start_vertex_idx_tuple) << ")\n";

  // allocate memory
  logFile << "\tAllocate row_pointers ...\n"
  row_pointers_0_to_1 = (eidType *)malloc((std::get<0>(partition_num_vertices_tuple) + 1) * sizeof(eidType));
  for (eidType u = 0; u < std::get<0>(partition_num_vertices_tuple) + 1; u++)
    row_pointers_0_to_1[u] = 0;
  row_pointers_0_to_2 = (eidType *)malloc((std::get<0>(partition_num_vertices_tuple) + 1) * sizeof(eidType));
  for (eidType u = 0; u < std::get<0>(partition_num_vertices_tuple) + 1; u++)
    row_pointers_0_to_2[u] = 0;
  row_pointers_1_to_2 = (eidType *)malloc((std::get<1>(partition_num_vertices_tuple) + 1) * sizeof(eidType));
  for (eidType u = 0; u < std::get<1>(partition_num_vertices_tuple) + 1; u++)
    row_pointers_1_to_2[u] = 0;
  
  // finish initialization and exit
  logFile << "Initialization finished!\n";
}

// destroy the subgraph
void Type3SubGraph::destroy()
{
  logFile << "Start destroying Type3SubGraph " << this_type3_subgraph_idx << " ...\n";

  // free the allocated memory
  logFile << "\tFree the allocated memory ...\n";
  free(edges_0_to_1);
  free(row_pointers_0_to_1);
  free(edges_0_to_2);
  free(row_pointers_0_to_2);
  free(edges_1_to_2);
  free(row_pointers_1_to_2);
  logFile << "\tAllocated memory freed!\n";

  // close the logFile
  logFile << "Gracefully finishing ...\n";
  logFile.close();
}

// add (u, v) to the Type3SubGraph (u < v is guaranteed)
void Type3SubGraph::add_edge(vidType u, int u_partition_idx, vidType v, int v_partition_idx)
{
  directedEdge edge(u, v);
  // there are 3 cases if u < v is guaranteed
  if (u_partition_idx == std::get<0>(this_subgraph_tuple) && v_partition_idx == std::get<1>(this_subgraph_tuple))
  {
    temp_edges_0_to_1.push_back(edge);
    row_pointers_0_to_1[u - std::get<0>(partition_start_vertex_idx_tuple)]++;
  }
  else if (u_partition_idx == std::get<0>(this_subgraph_tuple) && v_partition_idx == std::get<2>(this_subgraph_tuple))
  {
    temp_edges_0_to_2.push_back(edge);
    row_pointers_0_to_2[u - std::get<0>(partition_start_vertex_idx_tuple)]++;
  }
  else
  {
    temp_edges_1_to_2.push_back(edge);
    row_pointers_1_to_2[u - std::get<1>(partition_start_vertex_idx_tuple)]++;
  }
}

// reorder and create the CSR format
void Type3SubGraph::recude()
{
  // allocate memory for 'edges'
  edges_0_to_1 = (vidType *)malloc(temp_edges_0_to_1.size() * sizeof(vidType));
  edges_0_to_2 = (vidType *)malloc(temp_edges_0_to_2.size() * sizeof(vidType));
  edges_1_to_2 = (vidType *)malloc(temp_edges_1_to_2.size() * sizeof(vidType));

  // perform scanning on row_pointers
  for (vidType counter_idx = 0; counter_idx < std::get<0>(partition_num_vertices_tuple); counter_idx++)
  {
    row_pointers_0_to_1[counter_idx + 1] += row_pointers_0_to_1[counter_idx];
    row_pointers_0_to_2[counter_idx + 1] += row_pointers_0_to_2[counter_idx];
  }
  for (vidType counter_idx = 0; counter_idx < std::get<1>(partition_num_vertices_tuple); counter_idx++)
    row_pointers_1_to_2[counter_idx + 1] += row_pointers_1_to_2[counter_idx];

  // pop the edges from back one by one and place it to the correct place
  vidType start_idx = std::get<0>(partition_start_vertex_idx_tuple);
  while (temp_edges_0_to_1.size() > 0)
  {
    directedEdge edge = temp_edges_0_to_1[temp_edges_0_to_1.size() - 1];
    edges_0_to_1[--row_pointers_0_to_1[edge.first - start_idx]] = edge.second;
    temp_edges_0_to_1.pop_back();
  }
  while (temp_edges_0_to_2.size() > 0)
  {
    directedEdge edge = temp_edges_0_to_2[temp_edges_0_to_2.size() - 1];
    edges_0_to_2[--row_pointers_0_to_2[edge.first - start_idx]] = edge.second;
    temp_edges_0_to_2.pop_back();
  }
  start_idx = std::get<1>(partition_start_vertex_idx_tuple);
  while (temp_edges_1_to_2.size() > 0)
  {
    directedEdge edge = temp_edges_1_to_2[temp_edges_1_to_2.size() - 1];
    edges_1_to_2[--row_pointers_1_to_2[edge.first - start_idx]] = edge.second;
    temp_edges_1_to_2.pop_back();
  }

  // pull up the 'creation_finished' flag
  creation_finished = 1;

  // sort and print the result into the logFile
  logFile << "Type3SubGraph " << this_type3_subgraph_idx << " has finished reducing!\n";

  logFile << "\t" << std::get<0>(this_subgraph_tuple) << " to " << std::get<1>(this_subgraph_tuple) << ":\n";
  vidType partition_start_vertex_idx_0 = std::get<0>(partition_start_vertex_idx_tuple);
  vidType partition_num_vertices_0 = std::get<0>(partition_num_vertices_tuple);
  for (vidType u = partition_start_vertex_idx_0; u < partition_start_vertex_idx_0 + partition_num_vertices_0; u++)
  {
    std::sort(edges_0_to_1 + row_pointers_0_to_1[u - partition_start_vertex_idx_0], edges_0_to_1 + row_pointers_0_to_1[u + 1 - partition_start_vertex_idx_0]);
    logFile << "\t\t" << u << ": ";
    vidType u_0_to_1_deg = get_out_degree_0_to_1(u);
    for (vidType v_idx = 0; v_idx < u_0_to_1_deg; v_idx++)
      logFile << N_0_to_1(u, v_idx) << " ";
    logFile << "\n";
  }

  logFile << "\t" << std::get<0>(this_subgraph_tuple) << " to " << std::get<2>(this_subgraph_tuple) << ":\n";
  for (vidType u = partition_start_vertex_idx_0; u < partition_start_vertex_idx_0 + partition_num_vertices_0; u++)
  {
    std::sort(edges_0_to_2 + row_pointers_0_to_2[u - partition_start_vertex_idx_0], edges_0_to_2 + row_pointers_0_to_2[u + 1 - partition_start_vertex_idx_0]);
    logFile << "\t\t" << u << ": ";
    vidType u_0_to_2_deg = get_out_degree_0_to_2(u);
    for (vidType w_idx = 0; w_idx < u_0_to_2_deg; w_idx++)
      logFile << N_0_to_2(u, w_idx) << " ";
    logFile << "\n";
  }

  logFile << "\t" << std::get<1>(this_subgraph_tuple) << " to " << std::get<2>(this_subgraph_tuple) << ":\n";
  vidType partition_start_vertex_idx_1 = std::get<1>(partition_start_vertex_idx_tuple);
  vidType partition_num_vertices_1 = std::get<1>(partition_num_vertices_tuple);
  for (vidType v = partition_start_vertex_idx_1; v < partition_start_vertex_idx_1 + partition_num_vertices_1; v++)
  {
    std::sort(edges_1_to_2 + row_pointers_1_to_2[v - partition_start_vertex_idx_1], edges_1_to_2 + row_pointers_1_to_2[v + 1 - partition_start_vertex_idx_1]);
    logFile << "\t\t" << v << ": ";
    vidType v_1_to_2_deg = get_out_degree_1_to_2(v);
    for (vidType w_idx = 0; w_idx < v_0_to_1_deg; w_idx++)
      logFile << N_1_to_2(v, w_idx) << " ";
    logFile << "\n";
  }
}

vidType Type3SubGraph::get_out_degree_0_to_1(vidType u)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'get_out_degree_0_to_1()' is called!\n";
    exit(-1);
  }
  vidType start_vertex_idx = std::get<0>(partition_start_vertex_idx_tuple);
  return row_pointers_0_to_1[u - start_vertex_idx + 1] - row_pointers_0_to_1[u - start_vertex_idx];
}

vidType Type3SubGraph::get_out_degree_0_to_2(vidType u)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'get_out_degree_0_to_2()' is called!\n";
    exit(-1);
  }
  vidType start_vertex_idx = std::get<0>(partition_start_vertex_idx_tuple);
  return row_pointers_0_to_2[u - start_vertex_idx + 1] - row_pointers_0_to_2[u - start_vertex_idx];
}

vidType Type3SubGraph::get_out_degree_1_to_2(vidType v)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'get_out_degree_1_to_2()' is called!\n";
    exit(-1);
  }
  vidType start_vertex_idx = std::get<1>(partition_start_vertex_idx_tuple);
  return row_pointers_1_to_2[v - start_vertex_idx + 1] - row_pointers_1_to_2[v - start_vertex_idx];
}

vidType Type3SubGraph::N_0_to_1(vidType u, vidType n)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'N_0_to_1()' is called!\n";
    exit(-1);
  }
  vidType start_vertex_idx = std::get<0>(partition_start_vertex_idx_tuple);
  return edges_0_to_1[row_pointers_0_to_1[u - start_vertex_idx] + n];
}

vidType Type3SubGraph::N_0_to_2(vidType u, vidType n)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'N_0_to_2()' is called!\n";
    exit(-1);
  }
  vidType start_vertex_idx = std::get<0>(partition_start_vertex_idx_tuple);
  return edges_0_to_2[row_pointers_0_to_2[u - start_vertex_idx] + n];
}

vidType Type3SubGraph::N_1_to_2(vidType v, vidType n)
{
  // the function must be called after the creation of the subgraph has finished
  if (creation_finished == 0)
  {
    std::cout << "Error! Subgraph creation has not finished, but 'N_1_to_2()' is called!\n";
    exit(-1);
  }
  vidType start_vertex_idx = std::get<1>(partition_start_vertex_idx_tuple);
  return edges_1_to_2[row_pointers_1_to_2[v - start_vertex_idx] + n];
}

/**************************************** Definition of tool functions ***************************************/

// given the device_count, calculate how many Type3SubGraphs
int get_type3_subgraph_num(int device_count)
{
  return (device_count * (device_count - 1) * (device_count - 2)) / 6;
}

// given the global index of Type3SubGraph, calculate its 3-tuple
type3Tuple get_type3_subgraph_tuple(int this_type3_subgraph_idx, int num_partitions)
{
  // check whether the requirement is legal, in the legal range
  int type3_subgraph_num = get_type3_subgraph_num(num_partitions);
  if (this_type3_subgraph_idx < 0 || this_type3_subgraph_idx >= type3_subgraph_num)
  {
    std::cout << "Error! 'this_type3_subgraph_idx' out of range in 'get_type3_subgraph_tuple()' call!\n";
    exit(-1);
  }

  // travers and attempt
  int first;
  int second;
  int third;
  type3Tuple tuple;
  for (first = 0; first < num_partitions - 2; first++)
  {
    int num_subgraphs_with_first = (num_partitions - first - 1) * (num_partitions - first - 2) / 2;
    if (num_subgraphs_with_first <= this_type3_subgraph_idx)
      this_type3_subgraph_idx -= num_subgraphs_with_first;
    else
    {
      std::get<0>(tuple) = first;
      for (second = first + 1; second < num_partitions - 1; second++)
      {
        int num_subgraphs_with_second = num_partitions - second - 1;
        if (num_subgraphs_with_second <= this_type3_subgraph_idx)
          this_type3_subgraph_idx -= num_subgraphs_with_second;
        else
        {
          std::get<1>(tuple) = second;
          third = second + 1 + this_type3_subgraph_idx;
          std::get<2>(tuple) = third;
          break;
        }
      }
      break;
    }
  }
  return tuple;
}

// given the device_count and three seperate partition index, calculate the global index of the Type3SubGraph
int get_type3_subgraph_idx(int device_count, vidType u, vidType v, vidType w)
{
  // sort the three vertices into increasing order
  sort_three_vertices(&u, &v, &w);

  // accumulate index
  int type3_subgraph_idx = 0;
  for (vidType ui = 0; ui < u; ui++)
    type3_subgraph_idx += (device_count - ui - 1) * (device_count - ui - 2) / 2;
  for (vidType vi = ui + 1; vi < v; vi++)
    type3_subgraph_idx += device_count - vi - 1;
  type3_subgraph_idx += w - v - 1;
  return type3_subgraph_idx;
}

// sort the three vertices into increasing order
void sort_three_vertices(vidType *u, vidType *v, vidType *w)
{
  if (*u > *v) vidType temp = *v, *v = *u, *u = temp;
  if (*v > *w) vidType temp = *w, *w = *v, *v = temp;
  if (*u > *v) vidType temp = *v, *v = *u, *u = temp;
}

/**************************************** Definition of TCSolver ***************************************/

void TCSolver(Graph &g, uint64_t &total, int n_gpus, int chunk_size) {
  // start TCSolver
  std::string file_name = "log.txt";
  std::ofstream logFile(file_name);
  if (!logFile)
  {
    std::cerr << "Cannot open " << file_name << "\n";
    exit(-1);
  }
  logFile << "TCSolver starts ...\n";

  // read important information out from the super graph
  logFile << "Reading input graph ...\n";
  vidType super_graph_vertex_num = g.V();
  logFile << "|V| = " << super_graph_vertex_num << "\n";

  // get the device_count of the system
  logFile << "Looking for devices ...\n";
  int device_count;
  CUDA_SAFE_CALL(hipGetDeviceCount(&device_count));
  logFile << "\t" << device_count << " devices available!\n";

  // creating the subgraphs
  logFile << "Creating the subgraphs ...\n";
  std::vector<Type1SubGraph> type1_subgraphs(device_count);
  for (int device_idx = 0; device_idx < device_count; device_idx++)
    type1_subgraphs[device_idx].init(g, device_count, device_idx);
  logFile << "\tAll the Type1SubGraphs created!\n";
  int type3_subgraph_num = get_type3_subgraph_num(device_count);
  std::vector<Type3SubGraph> type3_subgraphs(device_count);
  for (int type3_subgraph_idx = 0 type3_subgraph_idx < type3_subgraph_num; type3_subgraph_idx++)
    type3_subgraphs[type3_subgraph_idx].init(g, device_count, type3_subgraph_idx);
  logFile << "\tAll the Type3SubGraphs created!\n";

  // map
  logFile << "Start mapping ...\n";
  int normal_vertex_number_each_partition = (super_graph_vertex_num - 1) / device_count + 1;
  for (int device_idx = 0; device_idx < device_count; device_idx++)
  {
    logFile << "\tMapping part " << device_idx << " ...\n";
    vidType partition_start_vertex = normal_vertex_number_each_partition * device_idx;
    vidType partition_end_vertex = (partition_start_vertex + normal_vertex_number_each_partition > super_graph_vertex_num) ? super_graph_vertex_num : partition_start_vertex + normal_vertex_number_each_partition;
    for (vidType u = partition_start_vertex; u < partition_end_vertex; u++)
    {
      vidType u_deg = g.out_degree(u);
      for (vidType v_idx = 0; v_idx < u_deg; v_idx++)
      {
        vidType v = g.N(u, v_idx);
        int v_partition_idx = v / normal_vertex_number_each_partition;
        if (device_idx == v_partition_idx)  // inner edge
          type1_subgraphs[device_idx].add_edge(u, v);
        else  // cross edge
        {
          // First, record in their coresponding Type1SubGraphs
          type1_subgraphs[device_idx].add_edge(u, v);
          type1_subgraphs[v_partition_idx].add_edge(v, u);

          // Second, record into all the related Type3SubGraphs
          for (vidType w = 0; w < device_count; w++)
          {
            if (w != u && w != v)
            {
              int type3_subgraph_idx = get_type3_subgraph_idx(device_count, u, v, w);
              if (u < v)
                type3_subgraphs[type3_subgraph_idx].add_edge(u, device_idx, v, v_partition_idx);
              else
                type3_subgraphs[type3_subgraph_idx].add_edge(v, v_partition_idx, u, device_idx);
            }
          }
        }
      }
    }
  }
  logFile << "Finish mapping!\n";

  // reduce
  logFile << "Start reducing ...\n";
  for (int device_idx = 0; device_idx < device_count; device_idx++)
    type1_subgraphs[device_idx].reduce();
  for (int type3_subgraph_idx = 0; type3_subgraph_idx < type3_subgraph_num; type3_subgraph_idx++)
    type3_subgraphs[type3_subgraph_idx].reduce();
  logFile << "Finish reducing ...\n";

  // end and exit
  logFile << "Destoying Type1SubGraphs ...\n";
  for (int device_idx = 0; device_idx < device_count; device_idx++)
    type1_subgraphs[device_idx].destroy();
  logFile << "All the Type1SubGraphs have been destoyed!\n";
  logFile << "Destoying Type3SubGraphs ...\n";
  for (int type3_subgraph_idx = 0; type3_subgraph_idx < type3_subgraph_num; type3_subgraph_idx++)
    type3_subgraphs[type3_subgraph_idx].destroy();
  logFile << "All the Type3SubGraphs have been destoyed!\n";
  logFile << "Gracefully finishing ...\n";
  logFile.close();
}
