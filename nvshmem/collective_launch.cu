#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <nvshmem.h>
#include <nvshmemx.h>

#define CUDA_CHECK(stmt)                                                          \
    do {                                                                          \
        hipError_t result = (stmt);                                              \
        if (hipSuccess != result) {                                              \
            fprintf(stderr, "[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, \
                    hipGetErrorString(result));                                  \
            exit(-1);                                                             \
        }                                                                         \
    } while (0)

#define NVSHMEM_CHECK(stmt)                                                                \
    do {                                                                                   \
        int result = (stmt);                                                               \
        if (NVSHMEMX_SUCCESS != result) {                                                  \
            fprintf(stderr, "[%s:%d] nvshmem failed with error %d \n", __FILE__, __LINE__, \
                    result);                                                               \
            exit(-1);                                                                      \
        }                                                                                  \
    } while (0)

__global__ void reduce_ring(int *target, int mype, int npes) {
    int peer = (mype + 1) % npes;
    int lvalue = mype;

    for (int i = 1; i < npes; i++) {
        nvshmem_int_p(target, lvalue, peer);
        nvshmem_barrier_all();
        lvalue = *target + mype;
        nvshmem_barrier_all();
    }
}

int main(int c, char *v[]) {
    int mype, npes, mype_node;

    nvshmem_init();
    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);

    // application picks the device each PE will use
    CUDA_CHECK(hipSetDevice(mype_node));
    int *u = (int *)nvshmem_calloc(1, sizeof(int));
    int *h = (int *)calloc(1, sizeof(int));

    void *args[] = {&u, &mype, &npes};
    dim3 dimBlock(1);
    dim3 dimGrid(1);

    NVSHMEM_CHECK(
        nvshmemx_collective_launch((const void *)reduce_ring, dimGrid, dimBlock, args, 0, 0));
    CUDA_CHECK(hipDeviceSynchronize());

    hipMemcpy(h, u, sizeof(int), hipMemcpyDeviceToHost);
    printf("results on device [%d] is %d \n", mype, h[0]);

    nvshmem_free(u);
    free(h);
    nvshmem_finalize();

    return 0;
}