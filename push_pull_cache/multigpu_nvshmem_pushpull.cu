#include "hip/hip_runtime.h"
#include "graph.h"
#include "graph_partition.h"
#include "nvshmem.h"
#include "nvshmemx.h"
#include "cutil_subset.h"
#include "common.h"
#include "graph_gpu.h"
#include "cuda_launch_config.hpp"
#include <hipcub/hipcub.hpp>
typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;
#include "bs_warp_vertex_nvshmem.cuh"

#include <fstream>
#include <string>

#include "graph_nvshmem.h"
#include "cross_gpu_fifo.h"
#include "graph_cache.h"

int initialize_nvshmem() {
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    std::cout << "mype_node = " << mype_node << "\n";
    CUDA_SAFE_CALL(hipSetDevice(mype_node));
    return nvshmem_my_pe();
}

void calculate_kernel_size(size_t &nthreads, size_t &nblocks, size_t &nwarps) {
    nthreads = BLOCK_SIZE;

    nblocks = 65536;
    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
    int max_blocks_per_SM = maximum_residency(warp_vertex_nvshmem, nthreads, 0);
    std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
    size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
    nblocks = std::min(6*max_blocks, nblocks);

    std::cout << "CUDA triangle counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

    nwarps = WARPS_PER_BLOCK;
}


void TCSolver(Graph &g, uint64_t &total, int n_partitions, int chunk_size) {
    // get device information
    int ndevices;
    CUDA_SAFE_CALL(hipGetDeviceCount(&ndevices));
    if (ndevices < n_partitions) {
        std::cout << "Only " << ndevices << " GPUs available!\n";
        exit(1);
    } else ndevices = n_partitions;
    int npes = nvshmem_n_pes();
    int mype = nvshmem_my_pe();

    // calculate the data about the gpu kernel
    size_t nthreads;
    size_t nblocks;
    size_t nwarps;
    calculate_kernel_size(nthreads, nblocks, nwarps);
    size_t n_total_warps_in_a_grid = nwarps * nblocks;
    
    // tasks for the warps
    int n_recv_blocks = nblocks / RECVER_RATIO;
    int n_send_blocks = nblocks / SENDER_RATIO;
    int n_normal_blocks = nblocks - n_recv_blocks - n_send_blocks;
    std::cout << "n_normal_blocks = " << n_normal_blocks << "\n";
    std::cout << "n_sender_blocks = " << n_send_blocks << "\n";
    std::cout << "n_recver_blocks = " << n_recv_blocks << "\n";

    // prepare the data on the device
    GraphNVSHMEM d_graph(g, mype, n_total_warps_in_a_grid);
    CrossGPUFIFO d_fifo(mype, g.get_max_degree(), n_total_warps_in_a_grid, ndevices, n_normal_blocks * nwarps, n_send_blocks * nwarps);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    nvshmem_barrier_all();

    // launch kernel
    AccType h_count = 0;
    AccType *d_count = (AccType *)nvshmem_malloc(sizeof(AccType));
    CUDA_SAFE_CALL(hipMemcpy(d_count, &h_count, sizeof(AccType), hipMemcpyHostToDevice));
    Timer t;
    t.Start();
    warp_vertex_nvshmem<<<nblocks, nthreads>>>(d_count, d_graph, d_fifo, ndevices);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    t.Stop();
    
    // finalize
    CUDA_SAFE_CALL(hipMemcpy(&h_count, d_count, sizeof(AccType), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    nvshmem_barrier_all();
    std::cout << "runtime[gpu " << mype << "] = " << t.Seconds() << " sec\n";
    total = h_count;
    nvshmem_finalize();
}


//////////////////////////////////////////////////////////////////////////////////////


// void TCSolver(Graph &g, uint64_t &total, int n_partitions, int chunk_size) {
//     auto nv = g.V();                // number of vertices
//     auto md = g.get_max_degree();   // max degree
//     int ndevices;                   // how many devices
//     CUDA_SAFE_CALL(hipGetDeviceCount(&ndevices));
//     if (ndevices < n_partitions) {
//         std::cout << "Only " << ndevices << " GPUs available!\n";
//         exit(1);
//     } else ndevices = n_partitions;
//     vidType max_subg_nv = g.get_max_subg_nv();
//     eidType max_subg_ne = g.get_max_subg_ne();

//     int npes = nvshmem_n_pes();
//     int mype = nvshmem_my_pe();
//     nvshmem_barrier_all();

//     // calculate the numbers about the gpus
//     size_t nthreads = BLOCK_SIZE;
//     size_t nblocks = 65536;
//     hipDeviceProp_t deviceProp;
//     CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
//     int max_blocks_per_SM = maximum_residency(warp_vertex_nvshmem_local, nthreads, 0);
//     max_blocks_per_SM = std::max(max_blocks_per_SM, (int)maximum_residency(warp_vertex_nvshmem_global, nthreads, 0));
//     std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
//     size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
//     nblocks = std::min(max_blocks, nblocks);
//     std::cout << "CUDA triangle counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";
//     size_t nwarps = WARPS_PER_BLOCK;
//     nvshmem_barrier_all();

//     // calculation related to the messages
//     size_t num_warps_each_gpu_each_iteration = nwarps * nblocks;
//     std::cout << "num_warps_each_gpu_each_iteration = " << num_warps_each_gpu_each_iteration << "\n";
//     size_t message_valid_size = 4;
//     size_t message_u_size = sizeof(vidType);
//     size_t message_degree_size = sizeof(vidType);
//     size_t message_bitmask_size = ((md-1)/32+1) * 4;
//     size_t message_list_size = md * sizeof(vidType);
//     size_t single_message_size = message_valid_size + message_u_size + message_degree_size + message_bitmask_size + message_list_size;
//     std::cout << "single_message_size = " << single_message_size << "\n";
//     size_t message_buffer_size = (ndevices - 1) * num_warps_each_gpu_each_iteration * single_message_size;
//     std::cout << "message_buffer_size = " << message_buffer_size << "\n";
//     int num_iterations = (max_subg_nv-1)/num_warps_each_gpu_each_iteration+1;
//     std::cout << "max_subg_nv = " << max_subg_nv << "\n";
//     std::cout << "num_iterations = " << num_iterations << "\n"; 
//     CUDA_SAFE_CALL(hipDeviceSynchronize());
//     nvshmem_barrier_all();

//     // prepare for the graph on gpu
//     GraphGPU d_graph;
//     d_graph.init_nvshmem(g, mype, message_buffer_size);
//     d_graph.set_max_degree(md);
//     CUDA_SAFE_CALL(hipDeviceSynchronize());
//     nvshmem_barrier_all();

//     // launch kernel
//     Timer t;
//     t.Start();
//     AccType h_count = 0;
//     AccType *d_count = (AccType *)nvshmem_malloc(sizeof(AccType));
//     CUDA_SAFE_CALL(hipMemcpy(d_count, &h_count, sizeof(AccType), hipMemcpyHostToDevice));
//     std::cout << "PE[" << mype << "] Start kernel\n";
//     CUDA_SAFE_CALL(hipDeviceSynchronize());
//     nvshmem_barrier_all();

//     int u_local_id_start = 0;
//     int u_local_id_end = num_warps_each_gpu_each_iteration;
//     warp_clear_valid<<<nblocks, nthreads>>>(d_graph);
//     CUDA_SAFE_CALL(hipDeviceSynchronize());
//     nvshmem_barrier_all();
//     for (int iteration_id = 0; iteration_id < num_iterations; iteration_id++) {
//         warp_vertex_nvshmem_local<<<nblocks, nthreads>>>(d_count, d_graph, u_local_id_start, u_local_id_end);
//         CUDA_SAFE_CALL(hipDeviceSynchronize());
//         nvshmem_barrier_all();
//         warp_vertex_nvshmem_global<<<nblocks, nthreads>>>(d_count, d_graph);
//         CUDA_SAFE_CALL(hipDeviceSynchronize());
//         nvshmem_barrier_all();
//         u_local_id_start += num_warps_each_gpu_each_iteration;
//         u_local_id_end += num_warps_each_gpu_each_iteration;
//     }

//     CUDA_SAFE_CALL(hipMemcpy(&h_count, d_count, sizeof(AccType), hipMemcpyDeviceToHost));
//     t.Stop();
//     CUDA_SAFE_CALL(hipDeviceSynchronize());
//     nvshmem_barrier_all();
//     std::cout << "runtime[gpu " << mype << "] = " << t.Seconds() << " sec\n";
//     total = h_count;
    
//     nvshmem_finalize();
//     // logfile.close();
// }